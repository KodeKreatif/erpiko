#include "hip/hip_runtime.h"
#ifdef ENABLE_CUDA
#pragma GCC diagnostic push
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#pragma GCC diagnostic pop
#endif


#define SIZE 256
__global__ void setup_kernel(hiprandStateMRG32k3a *state, unsigned int seed) {
  int id = threadIdx.x + blockIdx.x * SIZE; 
  hiprand_init(seed, id, 0, &state[id]);
} 

__global__ void run_kernel(hiprandStateMRG32k3a *state, unsigned int *result) { 
  int id = threadIdx.x + blockIdx.x * SIZE;
  hiprandStateMRG32k3a localState = state[id];
  unsigned int x = hiprand(&localState); 
  while (x == 0) {
    x = hiprand(&localState); 
  }
  state[id] = localState; 
  result[id] = x; 
}


void setup_kernel_rng(hiprandStateMRG32k3a *devMRGStates, unsigned int seed) {
  setup_kernel<<<SIZE, SIZE>>>(devMRGStates, seed);
}

void run_kernel_rng(hiprandStateMRG32k3a *devMRGStates, unsigned int *devResults) {
  run_kernel<<<SIZE, SIZE>>>(devMRGStates, devResults);
}
